#include "hip/hip_runtime.h"
#include <cassert>
#include <gputils/cuda_utils.hpp>

#include "../include/gpu_mm2.hpp"
#include "../include/PlanIterator2.hpp"

using namespace std;
using namespace gputils;

namespace gpu_mm2 {
#if 0
}   // pacify editor auto-indent
#endif


// Helper function called by tod2map_kernel()
__device__ void update_shmem(float *shmem, int idec, int ira, int cell_idec, int cell_ira, float cos_2a, float sin_2a, float t)
{
    bool dec_in_cell = ((idec & ~63) == cell_idec);
    bool ra_in_cell = ((ira & ~63) == cell_ira);
    int s = ((idec & 63) << 6) | (ira & 63);

    // Warp divergence here
    if (dec_in_cell && ra_in_cell) {
	atomicAdd(shmem + s, t);
	atomicAdd(shmem + s + 64*64, t * cos_2a);
	atomicAdd(shmem + s + 2*64*64, t * sin_2a);
    }

    // FIXME is this a good idea?
    // __syncwarp();
}

template<int W, bool Debug>
__global__ void __launch_bounds__(32*W, 1)
tod2map4_kernel(
    float *map,                              // Shape (3, nypix, nxpix)   where axis 0 = {I,Q,U}
    const float *tod,                        // Shape (nsamp,)
    const float *xpointing,                  // Shape (3, ndet, nt)    where axis 0 = {px_dec, px_ra, alpha}
    const ulong *plan_mt,                    // See long comment above. Shape (plan_ncltod,)
    long nsamp,                              // Number of TOD samples (= detectors * times)
    int nypix,                               // Length of map declination axis
    int nxpix,                               // Length of map RA axis
    uint nmt,
    uint nmt_per_block)
{
    __shared__ float shmem[3*64*64];
    
    // Threadblock has shape (32,W), so threadIdx.x is the laneId, and threadIdx.y is the warpId.
    const uint laneId = threadIdx.x;
    const uint warpId = threadIdx.y;

    PlanIterator2<W,Debug> iterator(plan_mt, nmt, nmt_per_block);

    while (iterator.get_cell()) {
	uint icell = iterator.icell;
	int iy0_cell = (icell >> 10) << 6;
	int ix0_cell = (icell & ((1<<10) - 1)) << 6;
	
	// Zero shared memmory
	for (int s = 32*warpId + laneId; s < 3*64*64; s += 32*W)
	    shmem[s] = 0;
    
	__syncthreads();

	while (iterator.get_cl()) {
	    // Value of 'cltod' is the same on each thread.
	    int cltod = iterator.icl;
	    
	    long s = (long(cltod) << 5) + laneId;
	    float x = tod[s];
	    float px_dec = xpointing[s];
	    float px_ra = xpointing[s + nsamp];
	    float alpha = xpointing[s + 2*nsamp];
	    
	    float cos_2a = cosf(2.0f * alpha);
	    float sin_2a = sinf(2.0f * alpha);
	    
	    int idec = int(px_dec);
	    int ira = int(px_ra);
	    float ddec = px_dec - float(idec);
	    float dra = px_ra - float(ira);
	    
	    if (Debug) {
		assert(idec >= 0);
		assert(idec < nypix-1);
		assert(ira >= 0);
		assert(ira < nxpix-1);
	    }
	    
	    update_shmem(shmem, idec,   ira,   iy0_cell, ix0_cell, cos_2a, sin_2a, x * (1.0f-ddec) * (1.0f-dra));
	    update_shmem(shmem, idec,   ira+1, iy0_cell, ix0_cell, cos_2a, sin_2a, x * (1.0f-ddec) * (dra));
	    update_shmem(shmem, idec+1, ira,   iy0_cell, ix0_cell, cos_2a, sin_2a, x * (ddec) * (1.0f-dra));
	    update_shmem(shmem, idec+1, ira+1, iy0_cell, ix0_cell, cos_2a, sin_2a, x * (ddec) * (dra));	    
	}
    
	__syncthreads();
	
	// Shared -> global
	
	for (int y = warpId; y < 64; y += W) {
	    for (int x = laneId; x < 64; x += 32) {
		int ss = 64*y + x;                            // shared memory offset
		int sg = (iy0_cell+y)*nxpix + (ix0_cell+x);  // global memory offset
		
		float t = shmem[ss];
		if (!__reduce_or_sync(ALL_LANES, t != 0))
		    continue;
		
		atomicAdd(map + sg, t);
		atomicAdd(map + sg + nypix*nxpix, shmem[ss+64*64]);
		atomicAdd(map + sg + 2*nypix*nxpix, shmem[ss+2*64*64]);
	    }
	}

	__syncthreads();
    }
}


void launch_tod2map4(
    gputils::Array<float> &map,                  // Shape (3, nypix, nxpix)   where axis 0 = {I,Q,U}
    const gputils::Array<float> &tod,            // Shape (nsamp,)
    const gputils::Array<float> &xpointing,      // Shape (3, ndet, nt)    where axis 0 = {px_dec, px_ra, alpha}
    const gputils::Array<ulong> &plan_mt,        // Shape (plan_nmt,)
    int nmt_per_block)
{
    static constexpr int W = 16;
    static constexpr bool Debug = false;
    
    long nsamp, nypix, nxpix;
    
    check_tod_and_init_nsamp(tod, nsamp, "launch_tod2map4", true);        // on_gpu=true
    check_map_and_init_npix(map, nypix, nxpix, "launch_tod2map4", true);  // on_gpu=true
    check_xpointing(xpointing, nsamp, "launch_tod2map4", true);           // on_gpu
    
    xassert(plan_mt.ndim == 1);
    xassert(plan_mt.is_fully_contiguous());

    int nmt = plan_mt.size;
    int nblocks = (nmt + nmt_per_block - 1) / nmt_per_block;
    // cout << "XXX nmt_per_block=" << nmt_per_block << ", nblocks=" << nblocks << endl;
    
    tod2map4_kernel<W,Debug> <<< nblocks, {32,W} >>>
	(map.data, tod.data, xpointing.data, plan_mt.data, nsamp, nypix, nxpix, nmt, nmt_per_block);
    
    CUDA_PEEK("tod2map4_kernel");
}


}  // namespace gpu_mm
