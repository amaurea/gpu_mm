#include "hip/hip_runtime.h"
#include "../include/gpu_mm2.hpp"
#include "../include/PlanIterator2.hpp"

#include <cassert>
#include <gputils/cuda_utils.hpp>

using namespace gputils;

namespace gpu_mm2 {
#if 0
}   // pacify editor auto-indent
#endif


// The "pre-map2tod" kernel partially zeroes the TOD.
// Launch with threadIdx = { 32*W } (not {32,W}).

template<typename T>
__global__ void pre_map2tod_kernel(T *tod, const ulong *plan_mt, uint nmt, uint nmt_per_block)
{
    uint imt0 = (blockIdx.x) * nmt_per_block + threadIdx.x;
    uint imt1 = (blockIdx.x + 1) * nmt_per_block;
    imt1 = (imt1 < nmt) ? imt1 : nmt;
    imt1 = (imt1 + 31U) & ~31U;
    
    for (uint imt = imt0; imt < imt1; imt += blockDim.x) {
	uint i = (imt < nmt) ? imt : (nmt-1);
	ulong mt = plan_mt[i];
	
	uint mt0 = uint(mt >> 20);
	uint icl = mt0 & ((1U << 26) - 1);
	bool flag = (imt < nmt) && ((mt0 & (1U << 26)) != 0);
	uint mask = __ballot_sync(ALL_LANES, flag);

	for (uint lane = 0; lane < 32; lane++) {
	    if (mask & (1U << lane)) {
		uint iclz = __shfl_sync(ALL_LANES, icl, lane);
		uint s = (ulong(iclz) << 5) + (threadIdx.x & 31);
		tod[s] = 0;
	    }
	}
    }
}


template<typename T>
static void launch_pre_map2tod(T *tod, const ulong *plan_mt, int nmt)
{
    static constexpr int W = 4;  // warps per threadblock
    static constexpr int nmt_per_block = 1024;
    
    xassert(tod != nullptr);
    xassert(plan_mt != nullptr);
    xassert(nmt > 0);

    int nblocks = (nmt + nmt_per_block - 1) / nmt_per_block;

    pre_map2tod_kernel<T> <<< nblocks, 32*W >>>
	(tod, plan_mt, nmt, nmt_per_block);

    CUDA_PEEK("pre_map2tod kernel launch");
}


// -------------------------------------------------------------------------------------------------


template<typename T>
__device__ T eval_tqu(T *sp, int iy, int ix, int iy0_cell, int ix0_cell, T cos_2a, T sin_2a)
{
    bool y_in_cell = ((iy & ~63) == iy0_cell);
    bool x_in_cell = ((ix & ~63) == ix0_cell);
    int s = ((iy & 63) << 6) | (ix & 63);
    T ret = 0;
    
    // Warp divergence here
    // FIXME could remove it I think
    if (y_in_cell && x_in_cell) {
	ret = sp[s] + (cos_2a * sp[s+64*64]) + (sin_2a * sp[s+2*64*64]);
    }

    __syncwarp();
    return ret;
}


template<typename T, int W, bool Debug>
__global__ void map2tod2_kernel(T *tod, const T *map, const T *xpointing, const ulong *plan_mt,
				uint nsamp, int nypix, int nxpix, uint nmt, uint nmt_per_block)
{
    static constexpr T one = 1;
    static constexpr T two = 2;
    
    // 48 KB in single precision, 96 KB in double precision.
    // __shared__ T shmem[3*64*64];
    T *shmem = dtype<T>::get_shmem();

    if constexpr (Debug) {
	assert(blockDim.x == 32);
	assert(blockDim.y == W);
    }
    
    // Threadblock has shape (32,W), so threadIdx.x is the laneId, and threadIdx.y is the warpId.
    const uint laneId = threadIdx.x;
    const uint warpId = threadIdx.y;
    
    PlanIterator2<W,Debug> iterator(plan_mt, nmt, nmt_per_block);

    // Outer loop over map cells

    while (iterator.get_cell()) {
	uint icell = iterator.icell;
	uint iy0_cell = (icell >> 10) << 6;
	uint ix0_cell = (icell & ((1<<10) - 1)) << 6;
	
	// Shared -> global
	
	for (int y = warpId; y < 64; y += W) {
	    for (int x = laneId; x < 64; x += 32) {
		int ss = 64*y + x;                           // shared memory offset
		int sg = (iy0_cell+y)*nxpix + (ix0_cell+x);  // global memory offset

		shmem[ss] = map[sg];
		shmem[ss + 64*64] = map[sg + nypix*nxpix];
		shmem[ss + 2*64*64] = map[sg + 2*nypix*nxpix];
	    }
	}
	     
	__syncthreads();

	// Inner loop over TOD cache lines

	while (iterator.get_cl()) {
	    uint icl = iterator.icl;
	    uint s = (icl << 5) + laneId;  // FIXME 32-bit overflow

	    T ypix = xpointing[s];
	    T xpix = xpointing[s + nsamp];
	    T alpha = xpointing[s + 2*nsamp];

	    // FIXME add 'status' argument, and calls to range_check_{xpix,ypix}().
	    normalize_xpix(xpix, nxpix);   // defined in gpu_mm2_internals.hpp

	    int iy0, iy1, ix0, ix1;
	    quantize_ypix(iy0, iy1, ypix, nypix);  // defined in gpu_mm2_internals.hpp
	    quantize_xpix(ix0, ix1, xpix, nxpix);  // defined in gpu_mm2_internals.hpp

	    T dy = ypix - iy0;
	    T dx = xpix - ix0;
	    
	    T cos_2a, sin_2a;
	    dtype<T>::xsincos(two*alpha, &sin_2a, &cos_2a);

	    T t = eval_tqu(shmem, iy0, ix0, iy0_cell, ix0_cell, cos_2a, sin_2a) * (one-dy) * (one-dx);
	    t += eval_tqu(shmem, iy0, ix1, iy0_cell, ix0_cell, cos_2a, sin_2a) * (one-dy) * (dx);
	    t += eval_tqu(shmem, iy1, ix0, iy0_cell, ix0_cell, cos_2a, sin_2a) * (dy) * (one-dx);
	    t += eval_tqu(shmem, iy1, ix1, iy0_cell, ix0_cell, cos_2a, sin_2a) * (dy) * (dx);

	    if (iterator.sid > 0)
		atomicAdd(tod+s, t);
	    else
		tod[s] = t;
	}

	__syncthreads();
    }
}


template<typename T>
void launch_map2tod2(T *tod, const T *map, const T *xpointing, const ulong *plan_mt, 
		     long nsamp, long nypix, long nxpix, int nmt, int nmt_per_block, bool debug)
{
    static constexpr int W = 16;  // warps per threadblock

    check_nsamp(nsamp, "launch_map2tod2");
    check_nypix(nypix, "launch_map2tod2");
    check_nxpix(nxpix, "launch_map2tod2");
    
    xassert(nmt > 0);
    xassert(nmt_per_block > 0);
    xassert((nmt_per_block % 32) == 0);
    
    launch_pre_map2tod(tod, plan_mt, nmt);
    
    int nblocks = (nmt + nmt_per_block - 1) / nmt_per_block;
    int shmem_nbytes = 3 * 64 * 64 * sizeof(T);
    
    if (debug) {
	map2tod2_kernel<T,W,true> <<< nblocks, {32,W}, shmem_nbytes >>>
	    (tod, map, xpointing, plan_mt, nsamp, nypix, nxpix, nmt, nmt_per_block);
    }
    else {
	map2tod2_kernel<T,W,false> <<< nblocks, {32,W}, shmem_nbytes >>>
	    (tod, map, xpointing, plan_mt, nsamp, nypix, nxpix, nmt, nmt_per_block);
    }

    CUDA_PEEK("map2tod2 kernel launch");
}


#define INSTANTIATE(T) \
    template void launch_map2tod2(T *tod, const T *map, const T *xpointing, const ulong *plan_mt, \
				  long nsamp, long nypix, long nxpix, int nmt, int nmt_per_block, bool debug);

INSTANTIATE(float);
INSTANTIATE(double);


}  // namespace gpu_mm2
