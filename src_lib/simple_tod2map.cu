#include "hip/hip_runtime.h"
#include "../include/gpu_mm2.hpp"

#include <iostream>
#include <gputils/Array.hpp>
#include <gputils/cuda_utils.hpp>

using namespace std;
using namespace gputils;

namespace gpu_mm2 {
#if 0
}   // pacify editor auto-indent
#endif



// -------------------------------------------------------------------------------------------------
//
// Some boilerplate, used to support T=float and T=double with the same C++ template.

template<typename T> struct dtype {};

template<> struct dtype<float>
{
    // https://docs.nvidia.com/cuda/cuda-math-api/group__CUDA__MATH__SINGLE.html
    static __device__ void xsincos(float x, float *sptr, float *cptr) { sincosf(x, sptr, cptr); }
    static __device__ float *get_shmem() { extern __shared__ float shmem_f[]; return shmem_f; }
};


template<> struct dtype<double>
{
    // https://docs.nvidia.com/cuda/cuda-math-api/group__CUDA__MATH__DOUBLE.html
    static __device__ void xsincos(double x, double *sptr, double *cptr) { sincos(x, sptr, cptr); }
    static __device__ double *get_shmem() { extern __shared__ double shmem_d[]; return shmem_d; }
};


// -------------------------------------------------------------------------------------------------


template<typename T>
__device__ void analyze_ypix(int &iy0, int &iy1, T &dy, T ypix, int nypix)
{
    iy0 = int(ypix);
    iy0 = (iy0 >= 0) ? iy0 : 0;
    iy0 = (iy0 <= nypix-2) ? iy0 : (nypix-2);
    iy1 = iy0 + 1;
    dy = ypix - iy0;
}


template<typename T>
__device__ void analyze_xpix(int &ix0, int &ix1, T &dx, T xpix, int nxpix)
{
    // Wrap around
    xpix = (xpix >= 0) ? xpix : (xpix + nxpix);
    xpix = (xpix <= nxpix) ? xpix : (xpix - nxpix);

    ix0 = int(xpix);
    ix0 = (ix0 >= 0) ? ix0 : 0;
    ix0 = (ix0 <= nxpix-1) ? ix0 : (nxpix-1);
    ix1 = (ix0 < (nxpix-1)) ? (ix0+1) : 0;
    dx = xpix - ix0;
}


// -------------------------------------------------------------------------------------------------


template<typename T>
__device__ void add_tqu(T *map, int npix, T t, T q, T u, T w)
{
    atomicAdd(map, w*t);
    atomicAdd(map + npix, w*q);
    atomicAdd(map + 2*npix, w*u);
}


template<typename T>
__global__ void simple_tod2map_kernel(T *map, const T *tod, const T *xpointing, uint nsamp, int nypix, int nxpix, uint nsamp_per_block)
{
    static constexpr T one = 1;
    
    int npix = nypix * nxpix;
    uint s0 = blockIdx.x * nsamp_per_block;
    uint s1 = min(nsamp, s0 + nsamp_per_block);
    
    for (uint s = s0 + threadIdx.x; s < s1; s += blockDim.x) {
	T ypix = xpointing[s];
	T xpix = xpointing[s + nsamp];
	T alpha = xpointing[s + 2*nsamp];
	T t = tod[s];
	
	int iy0, iy1, ix0, ix1;
	T dy, dx, q, u;
	
	analyze_ypix(iy0, iy1, dy, ypix, nypix);
	analyze_xpix(ix0, ix1, dx, xpix, nxpix);
	dtype<T>::xsincos(alpha, &q, &u);
	q *= t;
	u *= t;

	add_tqu(map + iy0*nypix + ix0, npix, t, q, u, (one-dy) * (one-dx));
	add_tqu(map + iy0*nypix + ix1, npix, t, q, u, (one-dy) * (dx));
	add_tqu(map + iy1*nypix + ix0, npix, t, q, u, (dy) * (one-dx));
	add_tqu(map + iy1*nypix + ix1, npix, t, q, u, (dy) * (dx));
    }
}


template<typename T>
void launch_simple_tod2map(Array<T> &map, const Array<T> &tod, const Array<T> &xpointing)
{
    uint nsamp_t, nsamp_x;
    int nypix, nxpix;
    
    check_map(map, nypix, nxpix, "launch_simple_tod2map");
    check_tod(tod, nsamp_t, "launch_simple_tod2map");
    check_xpointing(xpointing, nsamp_x, "launch_simple_tod2map");
    
    assert(nsamp_t == nsamp_x);

    int nthreads_per_block = 128;
    int nsamp_per_block = 1024;
    int nblocks = (nsamp_t + nsamp_per_block - 1) / nsamp_per_block;
    
    simple_tod2map_kernel <<< nblocks, nthreads_per_block >>>
	(map.data, tod.data, xpointing.data, nsamp_t, nypix, nxpix, nsamp_per_block);

    CUDA_PEEK("simple_tod2map kernel launch");
}


#define INSTANTIATE(T) \
    template void launch_simple_tod2map(Array<T> &map, const Array<T> &tod, const Array<T> &xpointing)

INSTANTIATE(float);
INSTANTIATE(double);


}  // namespace gpu_mm2
