//nvcc -o libtime_cufft.so time_cufft.cu -shared -lcufft -Xcompiler -fPIC -lgomp

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <omp.h>
#include <hip/hip_complex.h>





void cufft_c2r(float *out, hipfftComplex *data, int len, int ntrans, int isodd)
{
  int nout=2*(len-1)-isodd;
  //float *out;
  //hipMalloc(&out,sizeof(float)*nout*ntrans);
  hipfftHandle plan;
  
  if (hipfftPlan1d(&plan,nout,HIPFFT_C2R, ntrans)!=HIPFFT_SUCCESS)
    fprintf(stderr,"Error planning dft\n");
  for (int i=0;i<20;i++) {
    hipDeviceSynchronize();
    double t1=omp_get_wtime();
    if (hipfftExecC2R(plan,data,out)!=HIPFFT_SUCCESS)
      fprintf(stderr,"Error executing dft\n");
    hipDeviceSynchronize();
    double t2=omp_get_wtime();
    printf("took %12.4g seconds to do fft.\n",t2-t1);
  }
  if (hipfftDestroy(plan)!= HIPFFT_SUCCESS)
    fprintf(stderr,"Error destroying plan.\n");
}

/*--------------------------------------------------------------------------------*/
void cufft_c2r_columns(float *out, hipfftComplex *data,int len, int ntrans, int isodd)
{
  int nout=2*(len-1)+isodd;
  hipfftHandle plan;
  int rank=1;
  int inembed[rank] = {ntrans};
  int onembed[rank]={ntrans};
  int istride=ntrans;
  int idist=1;
  int ostride=ntrans;
  int odist=1;
  if (hipfftPlanMany(&plan,rank,&nout,inembed,istride,idist,onembed,ostride,odist,HIPFFT_C2R,ntrans)!=HIPFFT_SUCCESS)
    fprintf(stderr,"Error planning DFT in c2r_columns.\n");
  if (hipfftExecC2R(plan,data,out)!=HIPFFT_SUCCESS)
    fprintf(stderr,"Error executing DFT in c2r_columns.\n");
  if (hipfftDestroy(plan)!=HIPFFT_SUCCESS)
    fprintf(stderr,"Error destroying plan in c2r_columns.\n");

}

/*--------------------------------------------------------------------------------*/
extern "C" {
void cufft_c2r_host(float *out, hipfftComplex *data, int n, int m, int isodd,int axis)
{
  float *dout;
  hipfftComplex *din;
  int nn;
  if (axis==0)
    nn=2*(n-1)+isodd;
  else
    nn=2*(m-1)+isodd;
  if (hipMalloc((void **)&din,sizeof(hipfftComplex)*n*m)!=hipSuccess)
    fprintf(stderr,"error in hipMalloc\n");
  if (hipMemcpy(din,data,n*m*sizeof(hipfftComplex),hipMemcpyHostToDevice)!=hipSuccess)
    fprintf(stderr,"Error copying data to device.\n");
  if (axis==0) {
    if (hipMalloc((void **)&dout,sizeof(float)*nn*m)!=hipSuccess)
      fprintf(stderr,"error in hipMalloc\n");
    cufft_c2r_columns(dout,din,n,m,isodd);
    //printf("copying %d %d\n",nn,m);
    if (hipMemcpy(out,dout,sizeof(float)*nn*m,hipMemcpyDeviceToHost)!=hipSuccess)
      fprintf(stderr,"Error copying result to host in c2r\n");
  }
  else {
    if (hipMalloc((void **)&dout,sizeof(float)*n*nn)!=hipSuccess)
      fprintf(stderr,"error in hipMalloc\n");
    cufft_c2r(dout,din,m,n,isodd);
    //printf("copying %d %d\n",n,nn);
    if (hipMemcpy(out,dout,sizeof(float)*nn*n,hipMemcpyDeviceToHost)!=hipSuccess)
      fprintf(stderr,"Error copying result to host in c2r\n");
  

  }
}
}

/*--------------------------------------------------------------------------------*/
void cufft_r2c(hipfftComplex *out, float *data, int len, int ntrans)
{
  //int nout=len/2+1;
  hipfftHandle plan;
  
  if (hipfftPlan1d(&plan,len,HIPFFT_R2C, ntrans)!=HIPFFT_SUCCESS)
    fprintf(stderr,"Error planning dft\n");
  for (int i=0;i<20;i++)
    {
      hipDeviceSynchronize();
      double t1=omp_get_wtime();
      if (hipfftExecR2C(plan,data,out)!=HIPFFT_SUCCESS)
	fprintf(stderr,"Error executing dft\n");
      hipDeviceSynchronize();
      double t2=omp_get_wtime();
      printf("r2c took %12.4g\n",t2-t1);
    }
  if (hipfftDestroy(plan)!= HIPFFT_SUCCESS)
    fprintf(stderr,"Error destroying plan.\n");
}

/*--------------------------------------------------------------------------------*/
void cufft_r2c_columns(hipfftComplex *out, float *data, int len, int ntrans)
{
  //int nout=len/2+1;
  //printf("performing %d transforms of length %d %d\n",ntrans,len,nout);

  hipfftHandle plan;
  int rank=1;
  int inembed[rank] = {len};
  int onembed[rank]={ntrans};
  int istride=ntrans;
  int idist=1;
  int ostride=ntrans;
  int odist=1;
  //if (hipfftPlanMany(&plan,1,&nout,&one,len,1,&one,nout,1,HIPFFT_R2C,ntrans)!=HIPFFT_SUCCESS)
  //if (hipfftPlanMany(&plan,rank,&len,inembed,len,1,onembed,nout,1,HIPFFT_R2C,ntrans)!=HIPFFT_SUCCESS)
  if (hipfftPlanMany(&plan,rank,&len,inembed,istride,idist,onembed,ostride,odist,HIPFFT_R2C,ntrans)!=HIPFFT_SUCCESS)
    fprintf(stderr,"Error planning DFT in r2c_columns.\n");
  if (hipfftExecR2C(plan,data,out)!=HIPFFT_SUCCESS)
    fprintf(stderr,"Error executing DFT in r2c_columns.\n");
  if (hipfftDestroy(plan)!=HIPFFT_SUCCESS)
    fprintf(stderr,"Error destroying plan in r2c_columns.\n");
  
}


/*--------------------------------------------------------------------------------*/

extern "C" {
void cufft_r2c_host(hipfftComplex *out, float *data, int n, int m, int axis)
{
  hipfftComplex *dout;
  float *din;
  int nn;
  if (axis==0)
    nn=n/2+1;
  else
    nn=m/2+1;
  if (hipMalloc((void **)&din,sizeof(float)*n*m)!=hipSuccess)
    fprintf(stderr,"error in hipMalloc\n");
  if (hipMemcpy(din,data,n*m*sizeof(float),hipMemcpyHostToDevice)!=hipSuccess)
    fprintf(stderr,"Error copying data to device.\n");
  if (axis==0) {
    if (hipMalloc((void **)&dout,sizeof(hipfftComplex)*nn*m)!=hipSuccess)
      fprintf(stderr,"error in hipMalloc\n");
    cufft_r2c_columns(dout,din,n,m);
    //printf("copying %d %d\n",nn,m);
    if (hipMemcpy(out,dout,sizeof(hipfftComplex)*nn*m,hipMemcpyDeviceToHost)!=hipSuccess)
      fprintf(stderr,"Error copying result to host in r2c\n");
  }
  else {
    if (hipMalloc((void **)&dout,sizeof(hipfftComplex)*n*nn)!=hipSuccess)
      fprintf(stderr,"error in hipMalloc\n");
    cufft_r2c(dout,din,m,n);
    //printf("copying %d %d\n",n,nn);
    if (hipMemcpy(out,dout,sizeof(hipfftComplex)*nn*n,hipMemcpyDeviceToHost)!=hipSuccess)
      fprintf(stderr,"Error copying result to host in r2c\n");
  
  }
}
}



/*================================================================================*/


#if 0

int main(int argc, char *argv[])
{
  printf("Hello world!\n");
  int ndet=1000;
  int nsamp=1<<18;
  printf("nsamp is %d\n",nsamp);

  float *fdat=(float *)malloc(sizeof(float)*ndet*nsamp);
  if (fdat!=NULL)
    printf("successfully malloced array on host.\n");

  float *ddat;
  if (hipMalloc((void **)&ddat,sizeof(float)*nsamp*ndet)!=hipSuccess)
    fprintf(stderr,"error in hipMalloc\n");
  hipComplex *dtrans;
  if (hipMalloc((void **)&dtrans,sizeof(hipComplex)*nsamp*ndet)!=hipSuccess)
    fprintf(stderr,"error in hipMalloc\n");

  
  
}
#endif
