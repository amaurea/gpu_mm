#include "hip/hip_runtime.h"
#include "../include/gpu_mm2.hpp"
#include "../include/gpu_mm2_internals.hpp"  // ALL_LANES
#include "../include/PlanIterator2.hpp"

#include <vector>
#include <iostream>
#include <algorithm> // std::sort
#include <gputils/cuda_utils.hpp>
#include <gputils/rand_utils.hpp>
#include <gputils/string_utils.hpp>

using namespace std;
using namespace gputils;

namespace gpu_mm2 {
#if 0
}   // pacify editor auto-indent
#endif


// This file just contains helper functions for testing!


template<typename T>
__device__ void assert_equal_within_warp(T x)
{
    uint mask = __match_any_sync(ALL_LANES, x);
    assert(mask == ALL_LANES);
}


__device__ void assert_equal_within_block(uint *sp, uint x)
{
    int laneId = threadIdx.x;
    int warpId = threadIdx.y;
    int W = blockDim.y;
    
    if (laneId == 0)
	sp[warpId] = x;

    __syncthreads();

    if (warpId == 0) {
	uint y = sp[min(laneId,W-1)];
	assert_equal_within_warp(y);
    }
    
    __syncthreads();	
}


template<int W>
__global__ void iterator2_test_kernel(ulong *plan_mt, uint nmt, uint nmt_per_block, int *out_mt_counts)
{
    __shared__ uint shmem[W];
	
    assert(blockDim.x == 32);
    assert(blockDim.y == W);

    int laneId = threadIdx.x;
    // int warpId = threadIdx.y;
    
    PlanIterator2<W,true> iterator(plan_mt, nmt, nmt_per_block);
    
    while (iterator.get_cell()) {
	uint icell = iterator.icell;
	assert(icell < (1U << 20));
	assert_equal_within_block(shmem, icell);

	for (;;) {
	    uint imt = iterator.imt_next;

	    if (!iterator.get_cl())
		break;

	    uint icl = iterator.icl;
	    assert(icl < (1U << 26));
	    assert(imt < nmt);
	    
	    assert_equal_within_warp(imt);
	    assert_equal_within_warp(icl);
	    assert(iterator.icell == icell);

	    ulong mt = plan_mt[imt];
	    uint mt_icell = uint(mt) & ((1U << 20) - 1);
	    uint mt_icl = uint(mt >> 20) & ((1U << 26) - 1);
	    
	    assert(icell == mt_icell);
	    assert(icl == mt_icl);

	    if (laneId == 0)
		atomicAdd(out_mt_counts + imt, 1);
	}

	// No __syncthreads() needed, since assert_equal_within_block() calls __syncthreads().
    }

    uint sentinel = 1U << 27;
    assert_equal_within_block(shmem, sentinel);
}


void test_plan_iterator2(const Array<ulong> &plan_mt, uint nmt_per_block, int warps_per_threadblock)
{
    xassert(plan_mt.ndim == 1);
    xassert(plan_mt.is_fully_contiguous());
    xassert(plan_mt.size > 0);
    xassert(nmt_per_block > 0);
    xassert((nmt_per_block % 32) == 0);

    Array<ulong> plan_mt_cpu = plan_mt.to_host();
    Array<ulong> plan_mt_gpu = plan_mt.to_gpu();
    
    long nmt = plan_mt_cpu.size;
    ulong *mt = plan_mt_cpu.data;

    // A little error checking on the plan
    
    for (long i = 1; i < nmt; i++) {
	uint icell0 = uint(mt[i-1]) & ((1<<20) - 1);
	uint icell1 = uint(mt[i]) & ((1<<20) - 1);
	xassert(icell0 <= icell1);
    }

    // Output array
    
    Array<int> mt_counts({nmt}, af_gpu | af_zero);
    
    // Launch kernel
    
    int nblocks = (nmt + nmt_per_block - 1) / nmt_per_block;
    
    if (warps_per_threadblock == 4)
	iterator2_test_kernel<4> <<< nblocks, {32,4} >>>
	    (plan_mt_gpu.data, nmt, nmt_per_block, mt_counts.data);
    else if (warps_per_threadblock == 8)
	iterator2_test_kernel<8> <<< nblocks, {32,8} >>>
	    (plan_mt_gpu.data, nmt, nmt_per_block, mt_counts.data);
    else if (warps_per_threadblock == 16)
	iterator2_test_kernel<16> <<< nblocks, {32,16} >>>
	    (plan_mt_gpu.data, nmt, nmt_per_block, mt_counts.data);
    else
	throw runtime_error("test_plan_iterator2: unsupported value of warps_per_threadblock");

    CUDA_PEEK("iterator2 test kernel launch");
    CUDA_CALL(hipDeviceSynchronize());

    // Check results
    
    mt_counts = mt_counts.to_host();
    
    for (long i = 0; i < nmt; i++)
	xassert(mt_counts.data[i] == 1);
}

}  // namespace gpu_mm2
