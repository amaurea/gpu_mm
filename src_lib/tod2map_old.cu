#include "hip/hip_runtime.h"
#include "../include/gpu_mm.hpp"
#include "../include/gpu_mm_internals.hpp"  // ALL_LANES
#include <gputils/cuda_utils.hpp>

using namespace gputils;

namespace gpu_mm {
#if 0
}   // pacify editor auto-indent
#endif


// -------------------------------------------------------------------------------------------------
//
// reference_tod2map()


// Helper function called by reference_tod2map()
inline void update_map(float *map, long ipix, long npix, float cos_2a, float sin_2a, float t)
{
    xassert((ipix >= 0) && (ipix < npix));
    
    map[ipix] += t;
    map[ipix+npix] += t * cos_2a;
    map[ipix+2*npix] += t * sin_2a;
}


void reference_tod2map(Array<float> &map, const Array<float> &tod, const Array<float> &xpointing)
{
    long nsamp, ndec, nra;
    check_tod_and_init_nsamp(tod, nsamp, "reference_tod2map", false);     // on_gpu=false
    check_map_and_init_npix(map, ndec, nra, "reference_tod2map", false);  // on_gpu=false
    check_xpointing(xpointing, nsamp, "reference_tod2map", false);        // on_gpu=false

    long npix = long(ndec) * long(nra);

    // No memset(out, ...) here, since we want to accumulate (not overwrite) output.
    
    for (long s = 0; s < nsamp; s++) {
	float x = tod.data[s];
	float px_dec = xpointing.data[s];
	float px_ra = xpointing.data[s + nsamp];
	float alpha = xpointing.data[s + 2*nsamp];
	
	float cos_2a = cosf(2*alpha);
	float sin_2a = sinf(2*alpha);

	int idec = int(px_dec);
	int ira = int(px_ra);
	float ddec = px_dec - float(idec);
	float dra = px_ra - float(ira);
	
	xassert(idec >= 0);
	xassert(idec < ndec-1);
	xassert(ira >= 0);
	xassert(ira < nra-1);
	
	long ipix = long(idec) * long(nra) + ira;

	update_map(map.data, ipix,       npix, cos_2a, sin_2a, x * (1.0-ddec) * (1.0-dra));
	update_map(map.data, ipix+1,     npix, cos_2a, sin_2a, x * (1.0-ddec) * (dra));
	update_map(map.data, ipix+nra,   npix, cos_2a, sin_2a, x * (ddec) * (1.0-dra));
	update_map(map.data, ipix+nra+1, npix, cos_2a, sin_2a, x * (ddec) * (dra));
    }
}



// -------------------------------------------------------------------------------------------------
//
// GPU tod2map


// Helper function called by tod2map_kernel()
__device__ void update_shmem(float *shmem, int idec, int ira, int cell_idec, int cell_ira, float cos_2a, float sin_2a, float t)
{
    bool dec_in_cell = ((idec & ~63) == cell_idec);
    bool ra_in_cell = ((ira & ~63) == cell_ira);
    int s = ((idec & 63) << 6) | (ira & 63);

    // Warp divergence here
    if (dec_in_cell && ra_in_cell) {
	atomicAdd(shmem + s, t);
	atomicAdd(shmem + s + 64*64, t * cos_2a);
	atomicAdd(shmem + s + 2*64*64, t * sin_2a);
    }

    // FIXME is this a good idea?
    // __syncwarp();
}


__global__ void old_tod2map_kernel(
    float *map,                              // Shape (3, ndec, nra)   where axis 0 = {I,Q,U}
    const float *tod,                        // Shape (ndet, nt)
    const float *xpointing,                  // Shape (3, ndet, nt)    where axis 0 = {px_dec, px_ra, alpha}
    const int *plan_cltod_list,              // See long comment above. Shape (plan_ncltod,)
    const int *plan_quadruples,              // See long comment above. Shape (plan_nquadruples, 4)
    long nsamp,                              // Number of TOD samples (= detectors * times)
    int ndec,                                // Length of map declination axis
    int nra)                                 // Length of map RA axis
{
    __shared__ float shmem[3*64*64];
    
    // Read quadruple for this block.
    // (After this, we don't need the 'plan_quadruples' pointer any more.)
    
    plan_quadruples += 4 * blockIdx.x;
    int cell_idec = plan_quadruples[0];  // divisible by 64
    int cell_ira = plan_quadruples[1];   // divisible by 64
    int icl_start = plan_quadruples[2];
    int icl_end = plan_quadruples[3];

    // Shift values of (plan_cltod_list, icl_start, icl_end), so that 0 <= icl_start < 32.
    // The values of (icl_start, icl_end) are the same on all threads.
    int icl_sbase = icl_start & ~31;
    plan_cltod_list += icl_sbase;
    icl_start -= icl_sbase;
    icl_end -= icl_sbase;

    // Shift map pointer to per-thread (not per-block) base location
    const int idec_base = cell_idec + (threadIdx.x >> 6);
    const int ira_base = cell_ira + (threadIdx.x & 63);
    map += long(idec_base) * long(nra) + ira_base;
        
    // Read global memory -> shared.
    // Assumes blockIdx.x is a multiple of 64.

    const long npix = long(ndec) * long(nra);    
    const int spix = (blockDim.x >> 6) * nra;  // Global memory "stride" in loop below
    	
    do {
	const float *m = map;
	for (int s = threadIdx.x; s < 64*64; s += blockDim.x) {
	    shmem[s] = m[0];
	    shmem[s + 64*64] = m[npix];
	    shmem[s + 2*64*64] = m[2*npix];
	    m += spix;
	}
    } while (0);
    
    __syncthreads();

    // Outer loop over batches of 32 TOD cache lines.
    // The value of 'icl_warp' is the same on each thread.
    
    const int laneId = threadIdx.x & 31;
    
    for (int icl_warp = (threadIdx.x & ~31); icl_warp < icl_end; icl_warp += blockDim.x) {
	// Value of 'cltod_outer' is different on each thread.
	int cltod_outer = plan_cltod_list[icl_warp + laneId];

	// Values of (icl0, icl1) are the same on each thread.
	int icl0 = max(icl_warp, icl_start);
	int icl1 = min(icl_warp+32, icl_end);
	
	// Inner loop over TOD cache lines ('cltod')
	// The value of 'icl' is the same on each thread.
	
	for (int icl = icl0; icl < icl1; icl++) {
	    // Value of 'cltod' is the same on each thread.
	    int cltod = __shfl_sync(ALL_LANES, cltod_outer, icl & 31);

	    // By convention, negative cltods are allowed, but ignored.
	    if (cltod < 0)
		continue;

	    long s = (long(cltod) << 5) + laneId;
	    float x = tod[s];
	    float px_dec = xpointing[s];
	    float px_ra = xpointing[s + nsamp];
	    float alpha = xpointing[s + 2*nsamp];

	    float cos_2a = cosf(2.0f * alpha);
	    float sin_2a = sinf(2.0f * alpha);

	    int idec = int(px_dec);
	    int ira = int(px_ra);
	    float ddec = px_dec - float(idec);
	    float dra = px_ra - float(ira);

	    // assert(idec >= 0);
	    // assert(idec < ndec-1);
	    // assert(ira >= 0);
	    // assert(ira < nra-1);	    

	    update_shmem(shmem, idec,   ira,   cell_idec, cell_ira, cos_2a, sin_2a, x * (1.0f-ddec) * (1.0f-dra));
	    update_shmem(shmem, idec,   ira+1, cell_idec, cell_ira, cos_2a, sin_2a, x * (1.0f-ddec) * (dra));
	    update_shmem(shmem, idec+1, ira,   cell_idec, cell_ira, cos_2a, sin_2a, x * (ddec) * (1.0f-dra));
	    update_shmem(shmem, idec+1, ira+1, cell_idec, cell_ira, cos_2a, sin_2a, x * (ddec) * (dra));	    
	}
    }
    
    __syncthreads();

    // Write shared memory -> global
    // Assumes blockIdx.x is a multiple of 64.
    
    do {
	float *m = map;
	for (int s = threadIdx.x; s < 64*64; s += blockDim.x) {
	    m[0] = shmem[s];
	    m[npix] = shmem[s + 64*64];
	    m[2*npix] = shmem[s + 2*64*64];
	    m += spix;
	}
    } while (0);
}


void launch_old_tod2map(
    gputils::Array<float> &map,                  // Shape (3, ndec, nra)   where axis 0 = {I,Q,U}
    const gputils::Array<float> &tod,            // Shape (ndet, nt)
    const gputils::Array<float> &xpointing,      // Shape (3, ndet, nt)    where axis 0 = {px_dec, px_ra, alpha}
    const gputils::Array<int> &plan_cltod_list,  // Shape (plan_ncltod,)
    const gputils::Array<int> &plan_quadruples)  // Shape (plan_nquadruples, 4)
{
    long nsamp, ndec, nra;
    check_tod_and_init_nsamp(tod, nsamp, "old_map2tod", true);     // on_gpu=true
    check_map_and_init_npix(map, ndec, nra, "old_map2tod", true);  // on_gpu=true
    check_xpointing(xpointing, nsamp, "old_map2tod", true);        // on_gpu=true
    
    int nblocks = plan_quadruples.shape[0];
    
    old_tod2map_kernel<<< nblocks, 512 >>>
	(map.data, tod.data, xpointing.data, plan_cltod_list.data, plan_quadruples.data, nsamp, ndec, nra);
    
    CUDA_PEEK("old_tod2map_kernel");
}


}  // namespace gpu_mm
