#include "hip/hip_runtime.h"
#include "../include/gpu_mm.hpp"
#include <cassert>

using namespace gputils;

namespace gpu_mm {
#if 0
}   // pacify editor auto-indent
#endif



static void _check_map2tod_args(float *tod, const float *map, const float *xpointing, int ndet, int nt, int ndec, int nra)
{
    assert(tod != nullptr);
    assert(map != nullptr);
    assert(xpointing != nullptr);
    
    assert(ndet > 0);
    assert(nt > 0);
    assert(ndec > 0);
    assert(nra > 0);

    assert((nt % 32) == 0);
    assert((ndec % 64) == 0);
    assert((nra % 64) == 0);
}


static void _check_map2tod_args(Array<float> &tod, const Array<float> &map, const Array<float> &xpointing)
{
    assert(tod.on_gpu());
    assert(tod.ndim == 2);
    assert(tod.is_fully_contiguous());
    
    assert(map.on_gpu());
    assert(map.ndim == 3);
    assert(map.shape[0] == 3);
    assert(map.is_fully_contiguous());
    
    assert(xpointing.on_gpu());
    assert(xpointing.ndim == 3);
    assert(xpointing.shape[0] == 3);
    assert(xpointing.shape[1] == tod.shape[0]);
    assert(xpointing.shape[2] == tod.shape[1]);
    assert(xpointing.is_fully_contiguous());
}


void reference_map2tod(float *tod, const float *map, const float *xpointing, int ndet, int nt, int ndec, int nra)
{
    _check_map2tod_args(tod, map, xpointing, ndet, nt, ndec, nra);
    
    long ns = long(ndet) * long(ndec);

    for (long s = 0; s < ns; s++) {
	float px_dec = xpointing[s];
	float px_ra = xpointing[s + ns];
	float alpha = xpointing[s + 2*ns];
	float cos_2a = cosf(2*alpha);
	float sin_2a = sinf(2*alpha);

	int idec = int(px_dec);
	int ira = int(px_ra);
	float ddec = px_dec - float(idec);
	float dra = px_ra - float(ira);
	
	assert(idec > 0);
	assert(idec < ndec-1);
	assert(ira > 0);
	assert(ira < nra-1);
	
	long ipix = idec*nra + ira;
	float out = 0.0;
	
	float w = (1.0-ddec) * (1.0-dra);
	out += w * map[ipix];
	out += w * cos_2a * map[ipix+ns];
	out += w * sin_2a * map[ipix+2*ns];

	w = (1.0-ddec) * (dra);
	out += w * map[ipix + 1];
	out += w * cos_2a * map[ipix+ns + 1];
	out += w * sin_2a * map[ipix+2*ns + 1];
	
	w = ddec * (1.0 - dra);
	out += w * map[ipix + nra];
	out += w * cos_2a * map[ipix+ns + nra];
	out += w * sin_2a * map[ipix+2*ns + nra];
	
	w = ddec * dra;
	out += w * map[ipix + nra+1];
	out += w * cos_2a * map[ipix+ns + nra+1];
	out += w * sin_2a * map[ipix+2*ns + nra+1];

	tod[s] = out;
    }
}


void reference_map2tod(Array<float> &tod, const Array<float> &map, const Array<float> &xpointing)
{
    _check_map2tod_args(tod, map, xpointing);
    
    reference_map2tod(tod.data, map.data, xpointing.data,
		      tod.shape[0], tod.shape[1], map.shape[1], map.shape[2]);
}


// -------------------------------------------------------------------------------------------------


// Number of blocks should be: 
//   ceil(ndet * nt / nt_per_block)

__global__ void map2tod_kernel(float *tod, const float *map, const float *xpointing,
			       int ndet, int nt, int ndec, int nra, int nt_per_block)
{
    long s0 = long(blockIdx.x) * long(nt_per_block);
    long ns = long(ndet) * long(nt);
    
    tod += s0;
    map += s0;
    xpointing += s0;

    int n = min(ns - s0, long(nt_per_block));
    
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
	float px_dec = xpointing[i];
	float px_ra = xpointing[i + ns];
	float alpha = xpointing[i + 2*ns];
	float cos_2a = cosf(2*alpha);
	float sin_2a = sinf(2*alpha);

	int idec = int(px_dec);
	int ira = int(px_ra);
	int ipix = idec*nra + ira;
	float ddec = px_dec - float(idec);
	float dra = px_ra - float(ira);
	float out = 0.0;

	float w = (1.0-ddec) * (1.0-dra);
	out += w * map[ipix];
	out += w * cos_2a * map[ipix+ns];
	out += w * sin_2a * map[ipix+2*ns];

	w = (1.0-ddec) * (dra);
	out += w * map[ipix + 1];
	out += w * cos_2a * map[ipix+ns + 1];
	out += w * sin_2a * map[ipix+2*ns + 1];
	
	w = ddec * (1.0 - dra);
	out += w * map[ipix + nra];
	out += w * cos_2a * map[ipix+ns + nra];
	out += w * sin_2a * map[ipix+2*ns + nra];
	
	w = ddec * dra;
	out += w * map[ipix + nra+1];
	out += w * cos_2a * map[ipix+ns + nra+1];
	out += w * sin_2a * map[ipix+2*ns + nra+1];

	tod[i] = out;
    }
}


void launch_map2tod(float *tod, const float *map, const float *xpointing,
		    int ndet, int nt, int ndec, int nra, hipStream_t stream,
		    int nthreads_per_block, int nt_per_block)
{
    _check_map2tod_args(tod, map, xpointing, ndet, nt, ndec, nra);
    
    assert(nthreads_per_block > 0);
    assert((nthreads_per_block % 32) == 0);
    assert(nthreads_per_block <= 1024);
    assert(nt_per_block > 0);

    int m = nt_per_block;
    long nblocks = (long(ndet) * long(nt) + m - 1) / m;
    assert(nblocks < (1L << 31));

    map2tod_kernel<<< nblocks, nthreads_per_block, 0, stream >>>
	(tod, map, xpointing, ndet, nt, ndec, nra, nt_per_block);
}


void launch_map2tod(Array<float> &tod, Array<float> &map, Array<float> &xpointing,
		    hipStream_t stream, int nthreads_per_block, int nt_per_block)
{
    _check_map2tod_args(tod, map, xpointing);
    
    launch_map2tod(tod.data, map.data, xpointing.data,
		   tod.shape[0], tod.shape[1], map.shape[1], map.shape[2],   // (ndet, nt, ndec, nra)
		   stream, nthreads_per_block, nt_per_block);
}


}  // namespace gpu_mm
