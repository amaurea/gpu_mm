#include "hip/hip_runtime.h"
#include "../include/gpu_mm.hpp"
#include <cassert>

using namespace gputils;

namespace gpu_mm {
#if 0
}   // pacify editor auto-indent
#endif


// -------------------------------------------------------------------------------------------------
//
// reference_tod2map(): slow single-threaded CPU tod2map, for testing.


// Helper function called by reference_tod2map()
inline void update_map(float *map, int ndec, int nra,         // map in global memory
		       int cell_idec, int cell_ira,           // base coords of current cell
		       int idec, int ira,                     // coords of data to be added
		       float cos_2a, float sin_2a, float t)   // data to be added
{
    bool dec_in_cell = (idec >= cell_idec) && (idec < cell_idec+64);
    bool ra_in_cell = (ira >= cell_ira) && (idec < cell_ira+64);

    if (!dec_in_cell || !ra_in_cell)
	return;

    long npix = long(ndec) * long(nra);
    
    map[idec*nra + ira] += t;
    map[idec*nra + ira + npix] += t * cos_2a;
    map[idec*nra + ira + 2*npix] += t * sin_2a;
}


void reference_tod2map(
    float *map,                            // shape (3, ndec, nra)   where axis 0 = {I,Q,U}
    const float *tod,                      // shape (ndet, nt)
    const float *xpointing,                // shape (3, ndet, nt)    where axis 0 = {px_dec, px_ra, alpha}
    const unsigned int *plan_cltod_list,   // shape (plan_ncltod,)
    const unsigned int *plan_quadruples,   // shape (plan_nquadruples, 4)
    int plan_ncltod,                       // defines length of plan_cltod_list[] array
    int plan_nquadruples,                  // defines length of plan_quadruples[] array
    int ndet,                              // number of detectors
    int nt,                                // number of time samples per detector
    int ndec,                              // nength of map declination axis
    int nra)                               // nength of map RA axis
{
    assert(map != nullptr);
    assert(tod != nullptr);
    assert(xpointing != nullptr);
    assert(plan_cltod_list != nullptr);
    assert(plan_quadruples != nullptr);

    assert(plan_nquadruples > 0);
    assert(ndet > 0);
    assert(nt > 0);
    assert(ndec > 0);
    assert(nra > 0);

    // These are limitations of "version 0" and may go away in the future.
    assert((ndec % 64) == 0);
    assert((nra % 64) == 0);
    assert((nt % 32) == 0);

    // A "sample" is a (detector, time) pair.
    long nsamp = long(ndet) * long(nt);

    // This version of tod2map() overwites the existing map.
    long npix = long(ndec) * long(nra);
    memset(map, 0, 3 * npix * sizeof(float));
    
    for (int q = 0; q < plan_nquadruples; q++) {
	int cell_idec = plan_quadruples[4*q];
	int cell_ira = plan_quadruples[4*q+1];
	int cltod_list_istart = plan_quadruples[4*q+2];
	int cltod_list_iend = plan_quadruples[4*q+3];
	
	assert(cell_idec >= 0);
	assert((cell_idec % 64) == 0);
	assert((cell_idec + 64) <= ndec);
	
	assert(cell_ira >= 0);
	assert((cell_ira % 64) == 0);
	assert((cell_ira + 64) <= ndec);

	// Note that we don't allow (cltod_list_istart == cltod_list_iend),
	// since this probably indicates a bug in plan creation.
	
	assert(cltod_list_istart >= 0);
	assert(cltod_list_istart < cltod_list_iend);
	assert(cltod_list_iend <= plan_ncltod);
	
	for (int c = cltod_list_istart; c < cltod_list_iend; c++) {
	    int cltod = plan_cltod_list[c];

	    // By convention, negative cltods are allowed, but ignored.
	    // (This is convenient in the GPU kernel.)
	    
	    if (cltod < 0)
		continue;

	    long samp0 = 32 * long(cltod);
	    assert(samp0 <= nsamp);

	    for (long s = samp0; s < samp0+32; s++) {
		float x = tod[s];
		float px_dec = xpointing[s];
		float px_ra = xpointing[s + nsamp];
		float alpha = xpointing[s + 2*nsamp];
		
		float cos_2a = cosf(2*alpha);
		float sin_2a = sinf(2*alpha);

		int idec = int(px_dec);
		int ira = int(px_ra);
		float ddec = px_dec - float(idec);
		float dra = px_ra - float(ira);

		update_map(map, ndec, nra, cell_idec, cell_ira, idec,   ira,   cos_2a, sin_2a, x * (1.0-ddec) * (1.0-dra));
		update_map(map, ndec, nra, cell_idec, cell_ira, idec,   ira+1, cos_2a, sin_2a, x * (1.0-ddec) * (dra));
		update_map(map, ndec, nra, cell_idec, cell_ira, idec+1, ira,   cos_2a, sin_2a, x * (ddec) * (1.0-dra));
		update_map(map, ndec, nra, cell_idec, cell_ira, idec+1, ira+1, cos_2a, sin_2a, x * (ddec) * (dra));
	    }
	}
    }
}


}  // namespace gpu_mm
