#include "../include/gpu_mm2.hpp"

#include <iostream>
#include <gputils/cuda_utils.hpp>
#include <gputils/time_utils.hpp>
#include <gputils/string_utils.hpp>

using namespace std;
using namespace gputils;
using namespace gpu_mm2;


template<typename T>
static void time_pointing_plan()
{
    long nsamp = 256*1024*1024;
    long nypix = 8*1024;
    long nxpix = 32*1024;
    
    double scan_speed = 0.5;    // pixels per TOD sample
    double total_drift = 1024;  // x-pixels

    ToyPointing<T> tp(nsamp, nypix, nxpix, scan_speed, total_drift);
    PointingPrePlan pp(tp.xpointing_gpu, nypix, nxpix);

    Array<unsigned char> buf({pp.plan_nbytes}, af_gpu);
    Array<unsigned char> tmp_buf({pp.plan_constructor_tmp_nbytes}, af_gpu);
    
    for (int i = 0; i < 20; i++) {
	struct timeval tv0 = get_time();

	PointingPlan p(pp, tp.xpointing_gpu, buf, tmp_buf);
	CUDA_CALL(hipDeviceSynchronize());
	
	double dt = time_since(tv0);
	cout << "PointingPlan<" << type_name<T>() << ">: " << dt << " seconds" << endl;
    }
}


int main(int argc, char **argv)
{
    time_pointing_plan<float>();
    // time_preplan<double>(); // FIXME blue
    return 0;
}
       
